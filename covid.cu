#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<time.h> 
#include<math.h>

typedef struct
{
	int x, y, day_infected, R, state;
	bool symptoms;
} person;

typedef struct
{
	person *list;
} people;

// all of the people
person *g_population=NULL;

// people stored by location
person ***g_world=NULL;

// Current width of world.
size_t g_worldWidth=0;

/// Current height of world.
size_t g_worldHeight=0;

size_t g_popSize = 0;

static inline void covid_initMaster(unsigned int pop_size, size_t world_width, size_t world_height) {
	g_worldWidth = world_width;
	g_worldHeight = world_height;
	g_popSize = pop_size;


	// 
	int world_area = g_worldHeight * g_worldWidth;
	float pop_density = g_popsize / world_area;
	int depth = ceil(pop_density)*5;


	hipMallocManaged((void**)&g_population, g_popSize*sizeof(person));
	person *world;
	hipMallocManaged((void**)world, g_worldHeight*g_worldWidth*depth*sizeof(person));
	g_world = (person (**)[depth]) world;

	for (int i=0; i<g_popSize; ++i) {
		int x = rand() % (g_worldWidth + 1);
		int y = rand() % (g_worldHeight + 1);

		g_population[i].x = x;
		g_population[i].y = y;
		g_population[i].day_infected = NULL;
		g_population[i].R = 0;
		g_population[i].sate = 0;
		g_population[i].symptoms = false;
	}

    // test commit

}

int main(int argc, char *argv[]) {
	unsigned int pop_size, world_width, world_height, infection_radius, infection_chance, days;

	pop_size = atoi(argv[1]);
	world_width = atoi(argv[2]);
	world_height = atoi(argv[3]);
	infection_radius = atoi(argv[4]);
	days = atoi (argv[4]);
	unsigned int timesteps = days*24;

	srand(time(0));
	covid_initMaster(pop_size, world_width, world_height);


}

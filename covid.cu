#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<time.h> 
#include<math.h>

typedef struct
{
	int x, y, day_infected, R, state;
	bool symptoms;
} person;

typedef struct
{
	person *list;
} people;

// all of the people
person *g_population=NULL;

// people stored by location
person ***g_world=NULL;

// Current width of world.
size_t g_worldWidth=0;

/// Current height of world.
size_t g_worldHeight=0;

size_t g_popSize = 0;

static inline void covid_initMaster(unsigned int pop_size, size_t world_width, size_t world_height) {
	g_worldWidth = world_width;
	g_worldHeight = world_height;
	g_popSize = pop_size;


	// 
	int world_area = g_worldHeight * g_worldWidth;
	float pop_density = g_popsize / world_area;
	int depth = ceil(pop_density)*5;


	hipMallocManaged((void**)&g_population, g_popSize*sizeof(person));
	person *world;
	hipMallocManaged((void**)world, g_worldHeight*g_worldWidth*depth*sizeof(person));
	g_world = (person (**)[depth]) world;



	for (int i <)
}

int main(int argc, char *argv[]) {
	unsigned int pop_size, world_width, world_height, infection_radius, infection_chance;

	srand(time(0));
	covid_initMaster(pop_size, world_width, world_height);


}
